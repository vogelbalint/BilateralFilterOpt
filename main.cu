
#include <opencv2/core/core.hpp>
#include <opencv2/imgcodecs/imgcodecs.hpp>
#include <stdio.h>
#include <string.h>

#include "hip/hip_runtime.h"
#include ""

#include "helperfunctions.h"

#include "kernel.h"
#include "constant.h"
#include "constsize.h"

#define MAX_RANGE_DIFF 255

int main(int argc, char** argv) 
{
	//Ha a help argumentummal ind�tjuk a programot, ismertetj�k a program m�k�d�s�t.
	if (argc == 2 && strcmp("help", argv[1]) == 0) {
		printHelpMessage(stdout);
		return 0;
	}
	//Megn�zz�k, hogy van-e megfelel� GPU
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		fprintf(stderr, "You don't have a CUDA capable GPU. Buy one! Sorry.\n");
		return NO_DEVICE_ERROR;
	}
	hipSetDevice(0);

	float sigma_s, sigma_r;		//a megfelel� Gauss f�ggv�nyek param�terei
	int r, threads;				//r: a spatial kernel sugara, threads: a blokkonk�nti thread-ek sz�ma adott dimenzi�ban

	int returnValue = readConfigParameters(argc, argv, sigma_s, sigma_r, r, threads);
	if (returnValue != 0) {
		return returnValue;
	}

	cv::Mat image;						//openCV f�ggv�nnyel olvassuk be a k�pet.
	image = cv::imread(argv[1], 0);		//beolvassuk a k�pet, 8 bit sz�rke�rnyalatoss� konvert�ljuk
	if (!image.data) {
		fprintf(stderr, "Could not open or find the input image\n\n");
		return NO_IMAGE_ERROR;
	}

	int width = image.cols, height = image.rows;	//a k�p adatai
	int imageSize = width * height;

	int rangeKernelSize = MAX_RANGE_DIFF * 2 + 1;	//=511
	
	unsigned char *d_inputImage = NULL;	//felszabad�thatunk egy adott pointert a freeEverything f�gggv�nnyel, ez�rt minden pointer nullra �ll�tunk.
	float *d_rangeKernel = NULL;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	//felt�ltj�k a konstans mem�ri�t
	if (!fillConstantMemory(r, sigma_s)) {
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return CONST_MEM_FILL_ERROR;
	}

	//Az �sszes haszn�lt device mem�ri�t lefoglaljuk. Ha hiba van, kil�p�nk.
	if (!doAllMallocs(d_inputImage, d_rangeKernel, imageSize, rangeKernelSize)) {
		fprintf(stderr, "hipMalloc failed!\n\n");
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return CUDA_MALLOC_ERROR;
	}

	//bem�soljuk a device-ra a k�pet
	if (hipMemcpy(d_inputImage, image.data, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n\n");
		freeEverything(d_inputImage, d_rangeKernel);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return CUDA_MEMCPY_ERROR;
	}

	//range kernel felt�lt�se
	createRangeKernel << <1, rangeKernelSize >> >(d_rangeKernel, sigma_r, MAX_RANGE_DIFF);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("Something went wrong during the execution of the createRangeKernel cuda kernel \n\n");
		printf("CUDA error: %s\n", hipGetErrorString(error));
		freeEverything(d_inputImage, d_rangeKernel);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return KERNEL_ERROR;
	}

	//a k�p sz�l�nek r sz�less�g� r�sz�t nem processz�ljuk. Ez tal�n elfogadhat�.
	//a processz�lt r�sz m�retei:
	int processedPartWidth = width - 2 * r;
	int processedPartHeight = height - 2 * r;

	//a block dimenzi�k megad�sa.
	//pl x ir�nyban: a legkisebb olyan eg�sz sz�m kell, ami threads-nek t�bbsz�r�se, de nagyobb processsdPartWidth-n�l:
	int blocksX = (processedPartWidth + threads - 1) / threads;
	int blocksY = (processedPartHeight + threads - 1) / threads;

	//a blokkonk�nt bem�solt k�pr�szlet m�retei:
	int imagePartWidth = threads + 2 * r;
	int imagePartSize = imagePartWidth * imagePartWidth;

	//A l�nyeg:
	bilateralFilter <<<dim3(blocksX, blocksY), dim3(threads, threads), imagePartSize * sizeof(unsigned char) + rangeKernelSize * sizeof(float) >>>
					(d_inputImage, d_rangeKernel, r, MAX_RANGE_DIFF, width, height);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("Something went wrong during the execution of th bilateral filter kernel\n\n");
		printf("CUDA error: %s\n", hipGetErrorString(error));
		freeEverything(d_inputImage, d_rangeKernel);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return KERNEL_ERROR;
	}

	//k�p m�sol�sa device to host
	if (hipMemcpy(image.data, d_inputImage, imageSize * sizeof(unsigned char), hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n\n");
		freeEverything(d_inputImage, d_rangeKernel);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		return CUDA_MEMCPY_ERROR;
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;		//M�rj�k, hogy mennyi ideig tartott a GPU specifikus utas�t�sok v�grehajt�sa.
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Time to generate: %3.1f ms\n"
		"with parameters: sigma_s = %3.1f, sigma_r = %3.1f, spatial kernel radius = %d, number of threads per block dim = %d\n\n",
		elapsedTime, sigma_s, sigma_r, r, threads);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	//k�p ment�se
	if (!cv::imwrite(argv[2], image)) {
		fprintf(stderr, "Failed to save the processed image.\n\n");
		freeEverything(d_inputImage, d_rangeKernel);
		return NO_IMAGE_ERROR;
	}

	freeEverything(d_inputImage, d_rangeKernel);
	return 0;	//csak akkor t�r�nk vissza 0-val, ha minden rendben ment
}